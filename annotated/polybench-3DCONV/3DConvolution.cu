/**
 * 3DConvolution.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <hip/hip_runtime.h>
#include <stdarg.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>
#include <unistd.h>

// define a small float value
#define SMALL_FLOAT_VAL 0.00000001f

double rtclock() {
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday(&Tp, &Tzp);
  if (stat != 0)
    printf("Error return from gettimeofday: %d", stat);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

float absVal(float a) {
  if (a < 0) {
    return (a * -1);
  } else {
    return a;
  }
}

float percentDiff(double val1, double val2) {
  if ((absVal(val1) < 0.01) && (absVal(val2) < 0.01)) {
    return 0.0f;
  }

  else {
    return 100.0f *
           (absVal(absVal(val1 - val2) / absVal(val1 + SMALL_FLOAT_VAL)));
  }
}

// define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.5

#define GPU_DEVICE 0

/* Problem size */
#define NI 256
#define NJ 256
#define NK 256

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;

void conv3D(DATA_TYPE *A, DATA_TYPE *B) {
  int i, j, k;
  DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

  c11 = +2;
  c21 = +5;
  c31 = -8;
  c12 = -3;
  c22 = +6;
  c32 = -9;
  c13 = +4;
  c23 = +7;
  c33 = +10;

  for (i = 1; i < NI - 1; ++i) // 0
  {
    for (j = 1; j < NJ - 1; ++j) // 1
    {
      for (k = 1; k < NK - 1; ++k) // 2
      {
        // printf("i:%d\nj:%d\nk:%d\n", i, j, k);
        B[i * (NK * NJ) + j * NK + k] =
            c11 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k - 1)] +
            c13 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k - 1)] +
            c21 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k - 1)] +
            c23 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k - 1)] +
            c31 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k - 1)] +
            c33 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k - 1)] +
            c12 * A[(i + 0) * (NK * NJ) + (j - 1) * NK + (k + 0)] +
            c22 * A[(i + 0) * (NK * NJ) + (j + 0) * NK + (k + 0)] +
            c32 * A[(i + 0) * (NK * NJ) + (j + 1) * NK + (k + 0)] +
            c11 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k + 1)] +
            c13 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k + 1)] +
            c21 * A[(i - 1) * (NK * NJ) + (j + 0) * NK + (k + 1)] +
            c23 * A[(i + 1) * (NK * NJ) + (j + 0) * NK + (k + 1)] +
            c31 * A[(i - 1) * (NK * NJ) + (j + 1) * NK + (k + 1)] +
            c33 * A[(i + 1) * (NK * NJ) + (j + 1) * NK + (k + 1)];
      }
    }
  }
}

void init(DATA_TYPE *A) {
  int i, j, k;

  for (i = 0; i < NI; ++i) {
    for (j = 0; j < NJ; ++j) {
      for (k = 0; k < NK; ++k) {
        A[i * (NK * NJ) + j * NK + k] = i % 12 + 2 * (j % 7) + 3 * (k % 13);
      }
    }
  }
}

void compareResults(DATA_TYPE *B, DATA_TYPE *B_outputFromGpu) {
  int i, j, k, fail;
  fail = 0;

  // Compare result from cpu and gpu...
  for (i = 1; i < NI - 1; ++i) // 0
  {
    for (j = 1; j < NJ - 1; ++j) // 1
    {
      for (k = 1; k < NK - 1; ++k) // 2
      {
        if (percentDiff(B[i * (NK * NJ) + j * NK + k],
                        B_outputFromGpu[i * (NK * NJ) + j * NK + k]) >
            PERCENT_DIFF_ERROR_THRESHOLD) {
          fail++;
        }
      }
    }
  }

  // Print results
  printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f "
         "Percent: %d\n",
         PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

void GPU_argv_init() {
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
  printf("setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice(GPU_DEVICE);
}

__global__ void
convolution3D_kernel(DATA_TYPE __attribute__((annotate("256,256,256"))) * A,
                     DATA_TYPE __attribute__((annotate("256,256,256"))) * B,
                     int __attribute__((annotate("4"))) i)
    __attribute__((annotate("32,8:8,32"))) {
  int k = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

  c11 = +2;
  c21 = +5;
  c31 = -8;
  c12 = -3;
  c22 = +6;
  c32 = -9;
  c13 = +4;
  c23 = +7;
  c33 = +10;

  if ((i < (NI - 1)) && (j < (NJ - 1)) && (k < (NK - 1)) && (i > 0) &&
      (j > 0) && (k > 0)) {
    B[i * (NK * NJ) + j * NK + k] =
        c11 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k - 1)] +
        c13 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k - 1)] +
        c21 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k - 1)] +
        c23 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k - 1)] +
        c31 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k - 1)] +
        c33 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k - 1)] +
        c12 * A[(i + 0) * (NK * NJ) + (j - 1) * NK + (k + 0)] +
        c22 * A[(i + 0) * (NK * NJ) + (j + 0) * NK + (k + 0)] +
        c32 * A[(i + 0) * (NK * NJ) + (j + 1) * NK + (k + 0)] +
        c11 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k + 1)] +
        c13 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k + 1)] +
        c21 * A[(i - 1) * (NK * NJ) + (j + 0) * NK + (k + 1)] +
        c23 * A[(i + 1) * (NK * NJ) + (j + 0) * NK + (k + 1)] +
        c31 * A[(i - 1) * (NK * NJ) + (j + 1) * NK + (k + 1)] +
        c33 * A[(i + 1) * (NK * NJ) + (j + 1) * NK + (k + 1)];
  }
}

void convolution3DCuda(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *B_outputFromGpu) {
  double t_start, t_end;

  DATA_TYPE *A_gpu;
  DATA_TYPE *B_gpu;

  hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NJ * NK);
  hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NI * NJ * NK);
  hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NJ * NK,
             hipMemcpyHostToDevice);
  hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NI * NJ * NK,
             hipMemcpyHostToDevice);

  dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid((size_t)(ceil(((float)NK) / ((float)block.x))),
            (size_t)(ceil(((float)NJ) / ((float)block.y))));

  t_start = rtclock();
  int i;
  for (i = 1; i < NI - 1; ++i) // 0
  {
    convolution3D_kernel<<<grid, block>>>(A_gpu, B_gpu, i);
  }

  hipDeviceSynchronize();
  t_end = rtclock();
  fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

  hipMemcpy(B_outputFromGpu, B_gpu, sizeof(DATA_TYPE) * NI * NJ * NK,
             hipMemcpyDeviceToHost);

  hipFree(A_gpu);
  hipFree(B_gpu);
}

int main(int argc, char *argv[]) {
  double t_start, t_end;

  DATA_TYPE *A;
  DATA_TYPE *B;
  DATA_TYPE *B_outputFromGpu;

  A = (DATA_TYPE *)malloc(NI * NJ * NK * sizeof(DATA_TYPE));
  B = (DATA_TYPE *)malloc(NI * NJ * NK * sizeof(DATA_TYPE));
  B_outputFromGpu = (DATA_TYPE *)malloc(NI * NJ * NK * sizeof(DATA_TYPE));

  init(A);

  GPU_argv_init();

  convolution3DCuda(A, B, B_outputFromGpu);

  t_start = rtclock();
  conv3D(A, B);
  t_end = rtclock();
  fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

  compareResults(B, B_outputFromGpu);

  free(A);
  free(B);
  free(B_outputFromGpu);

  return 0;
}